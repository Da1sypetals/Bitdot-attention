#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string>
#include <stdexcept>

__global__ void pack_bits_kernel(
    const bool* __restrict__ fb,   // (n, d)
    int32_t* __restrict__ o,       // (n * nc)
    int n, int d, int cb, int nc
) {
    int a = blockIdx.x * blockDim.x + threadIdx.x;   // 0 .. n*nc-1
    if (a >= n * nc) return;
    int b = a / nc;
    int c = a % nc;
    int e = c * cb;
    int f = min(e + cb, d);
    int32_t g = 0;
    for (int h = e; h < f; ++h) {
        int32_t i = (int32_t)(fb[b * d + h]);
        g |= (i << (h - e));
    }
    o[a] = g;
}

void pack_bits_cuda_launcher(
    const bool* f_binary,  // device ptr, shape (n, d_f)
    int32_t* out,             // device ptr, shape (n * n_chunks)
    int n, int d_f, int max_chunk_bits,
    hipStream_t stream = 0
) {
    const int n_chunks = (d_f + max_chunk_bits - 1) / max_chunk_bits;
    const int total_chunks = n * n_chunks;

    const int threads_per_block = 256;
    const int total_threads = total_chunks;
    const int num_blocks = (total_threads + threads_per_block - 1) / threads_per_block;

    pack_bits_kernel<<<num_blocks, threads_per_block, 0, stream>>>(
        f_binary, out, n, d_f, max_chunk_bits, n_chunks
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA error: ") + hipGetErrorString(err));
    }

}

